#include "hip/hip_runtime.h"
#include "cudaMethods.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <cmath>
#include <vector>
#include <iomanip>
#include <cassert>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 256

float dotProductCPU(float* a, float* b, int n) {
	float sum = 0;
	for (int i = 0; i < n; i++) {
		sum += a[i] * b[i];
	}
	return sum;
}

__global__ void dotProductGPU(float* a, float* b, float* result, int n) {
	extern __shared__ float sdata[];
	//���������� ������ ������ ������������ �����
	int tid = threadIdx.x + blockIdx.x * blockDim.x; //��������� ������ � ������� ����� + ������ ����� � ����� * ������ �����
																									//���������� ��������
	int local_tid = threadIdx.x;

	float temp = 0;


	if (tid < n)
		temp += a[tid] * b[tid];

	sdata[local_tid] = temp;
	__syncthreads();

	// ��������� ������������ �������� ������ �����
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (local_tid < s) {
			sdata[local_tid] += sdata[local_tid + s];
		}
		__syncthreads();
	}

	if (local_tid == 0) {
		atomicAdd(result, sdata[0]);
	}
}

//��������� ������ �� ���
void allocateGPUMemory(float** d_a, float** d_b, float** d_result, int N) {
	hipMalloc((void**)d_a, N * sizeof(float));
	hipMalloc((void**)d_b, N * sizeof(float));
	hipMalloc((void**)d_result, sizeof(float));
}

void copyToGPU(float* d_a, float* d_b, float* h_a, float* h_b, int N, float* d_result, float* h_result) {
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_result, h_result, sizeof(float), hipMemcpyHostToDevice);
}

void copyToCPU(float* h_result, float* d_result) {
	hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
}

void freeGPUMemory(float* d_a, float* d_b, float* d_result) {
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
}

float executeGPU(float* d_a, float* d_b, float* d_result, int N) {
	hipEvent_t startGPU, stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);

	hipEventRecord(startGPU);

	int blockSize = BLOCKSIZE;
	int gridSize = (N + blockSize - 1) / blockSize;

	dotProductGPU <<< gridSize, blockSize >>> (d_a, d_b, d_result, N);
	hipEventRecord(stopGPU);

	hipEventSynchronize(stopGPU);
	float gpuTime = 0.0f;
	hipEventElapsedTime(&gpuTime, startGPU, stopGPU);

	hipEventDestroy(startGPU);
	hipEventDestroy(stopGPU);

	return gpuTime;
}



CPUResult execute�PU(float* h_a, float* h_b, int N) {
	auto startCPU = std::chrono::high_resolution_clock::now();
	float cpuResult = dotProductCPU(h_a, h_b, N);
	auto endCPU = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float, std::milli> durationCPU = endCPU - startCPU;

	CPUResult result;
	result.result = cpuResult;
	result.duration = durationCPU.count();

	return result;
}
